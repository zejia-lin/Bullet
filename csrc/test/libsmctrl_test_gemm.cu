#include "hip/hip_runtime.h"
#include <errno.h>
#include <error.h>
#include <stdbool.h>
#include <stdio.h>

#include <iostream>

#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "check_cuda.h"
#include "libsmctrl.h"

#define SAFE(x) x

__global__ void helloworld() {
  printf("hello world");
  printf("(%d,%d) ", threadIdx.x, blockIdx.x);
  // if (threadIdx.x != 1)
  //   return;
  // int smIdx;
  // asm("mov.u32 %0, %%smid;"
  //     : "=r"(smIdx));
  // printf("%d, ", smIdx);
}

int main() {
  const int NUM_BLOCKS = 400;
  int res;
  uint32_t num_tpcs;
  int num_sms, sms_per_tpc;

  // Determine number of SMs per TPC
  checkCuda(hipDeviceGetAttribute(&num_sms, hipDeviceAttributeMultiprocessorCount, 0));
  if (res = libsmctrl_get_tpc_info_cuda(&num_tpcs, 0))
    error(1, res, "libsmctrl_test_global: Unable to get TPC configuration for test");
  sms_per_tpc = num_sms / num_tpcs;

  printf("Find %d SM, %d SM/TPC\n", num_sms, sms_per_tpc);

  // Test baseline (native) behavior without partitioning
  // printf("Before partition\n");
  // helloworld<<<1, 1>>>();
  // checkCuda(hipDeviceSynchronize());
  // helloworld<<<NUM_BLOCKS, 128>>>();
  // checkCuda(hipDeviceSynchronize());

  // printf("\nAfter partition\n");
  // libsmctrl_set_global_mask(~0x3ull);
  // helloworld<<<NUM_BLOCKS, 128>>>();
  // checkCuda(hipDeviceSynchronize());
  // exit(0);

  const size_t MAX_PROMPTS = 5120;
  const size_t MAX_DMODEL = 5120;
  const size_t MLP_HIDDEN = 22016;
  const int NUM_RUNS = 5;
  size_t *X, *W, *Y;
  const float ONE = 1;
  hipEvent_t st, ed;
  float duration;
  hipMalloc(&X, sizeof(__half) * MLP_HIDDEN * MLP_HIDDEN);
  hipMalloc(&W, sizeof(__half) * MLP_HIDDEN * MLP_HIDDEN);
  hipMalloc(&Y, sizeof(float) * MLP_HIDDEN * MLP_HIDDEN);
  checkCuda(hipEventCreate(&st));
  checkCuda(hipEventCreate(&ed));

  std::cout << "num_prompts,tpcs,time\n";
  for (size_t num_promts = 4096 - 8; num_promts <= 4096 + 8; num_promts += 1) {
    size_t d_model = 4096;
    size_t M = num_promts;
    size_t K = d_model;
    size_t N = 22016;

    hipblasHandle_t blasHandle;
    hipblasOperation_t trans = HIPBLAS_OP_N;
    checkCuda(hipblasCreate(&blasHandle));
    for (int i = 0; i < NUM_RUNS; ++i) {
      checkCuda(hipblasGemmEx(blasHandle, trans, trans,
                             M, N, K,
                             &ONE, X, HIP_R_16F, M,
                             W, HIP_R_16F, K,
                             &ONE, Y, HIP_R_32F, M,
                             HIPBLAS_COMPUTE_32F, HIPBLAS_GEMM_DEFAULT));
    }
    checkCuda(hipDeviceSynchronize());

    hipStream_t stream;
    checkCuda(hipStreamCreate(&stream));
    checkCuda(hipblasSetStream(blasHandle, stream));

    unsigned long long tpcs = 0ULL;
    for (int _ = 0; _ < 54; ++_) {
      tpcs = (tpcs << 1) | 1ULL;
      if(_ < 53){
        // continue;
      }
      libsmctrl_set_stream_mask(stream, ~tpcs);
      // libsmctrl_set_global_mask(~tpcs);
      // checkCuda(cublasSetSmCountTarget(blasHandle, 32));
      checkCuda(hipEventRecord(st));
      for (int i = 0; i < NUM_RUNS; ++i) {
        checkCuda(hipblasGemmEx(blasHandle, trans, trans,
                               M, N, K,
                               &ONE, X, HIP_R_16F, M,
                               W, HIP_R_16F, K,
                               &ONE, Y, HIP_R_32F, M,
                               HIPBLAS_COMPUTE_32F, HIPBLAS_GEMM_DEFAULT));
      }
      checkCuda(hipDeviceSynchronize());
      checkCuda(hipEventRecord(ed));
      checkCuda(hipDeviceSynchronize());
      checkCuda(hipEventElapsedTime(&duration, st, ed));
      std::cout << num_promts << "," << __builtin_popcountll(tpcs) << "," << duration / NUM_RUNS << "\n";
    }
  }
}
