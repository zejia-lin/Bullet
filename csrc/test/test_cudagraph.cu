#include "hip/hip_runtime.h"
#include <cstdio>

#include <iostream>

#include "check_cuda.h"
#include "libsmctrl.h"
#include "libsmctrl_test_mask_shared.h"

__global__ void echo_sm(int *used_sm) {
  if (threadIdx.x != 1)
    return;
  int smIdx;
  asm("mov.u32 %0, %%smid;"
      : "=r"(smIdx));
  printf("%d, ", smIdx);
  used_sm[smIdx] = 1;
}

int main(int argc, char **argv) {
  int *used_sm;
  int num_sms;
  uint64_t mask;

  hipGraph_t graph;
  hipGraphExec_t instance;
  hipStream_t stream, s2;
  checkCuda(hipStreamCreate(&stream));
  checkCuda(hipStreamCreate(&s2));

  checkCuda(hipDeviceGetAttribute(&num_sms, hipDeviceAttributeMultiprocessorCount, 0));
  checkCuda(hipMallocManaged(&used_sm, sizeof(int) * num_sms));

  int low = atoi(argv[1]);
  int high = atoi(argv[2]);
  libsmctrl_make_mask(&mask, low, high);
  libsmctrl_set_stream_mask(s2, mask);
  // libsmctrl_set_global_mask(mask);

  hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
  for (int i = 0; i < 5; ++i) {
    echo_sm<<<256, 12, 0, stream>>>(used_sm);
  }
  hipStreamEndCapture(stream, &graph);
  hipGraphInstantiate(&instance, graph, NULL, NULL, 0);
  hipGraphLaunch(instance, s2);
  checkCuda(hipDeviceSynchronize());
  
  for (int i = 0; i < num_sms; ++i) {
    if (used_sm[i] == 1) {
      if ((i / 2 < low) or (i / 2 >= high)) {
        std::cout << "SM " << i << " shouldn't be used\n";
      }
    } else if ((low <= i / 2) && (i / 2 < high)) {
      std::cout << "SM " << i << " should be used\n";
    }
  }
}